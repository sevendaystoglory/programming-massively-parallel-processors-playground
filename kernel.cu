#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#define N 1 << 20 // 2^28 ~ 268M elements. ~3 GiB for 3 arrays.

// we shall understand thread hierarchy
__global__ void VecAdd(int* A, int* B, int * C){
    int start = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = start; i<N; i+=blockDim.x * gridDim.x){ // for a lower grid dim as imposed by the min, we have a longer running kernel
        C[i] = A[i] + B[i];    
    }
}

void print_array(int* A, int* B, int* C);
void seeCUDAerror();

int main(){
    // initialize some variables in CPU memory

    const int size = N;
    int* A = (int*)malloc(size * sizeof(int)); // allocate memory in heap as really large.
    int* B = (int*)malloc(size * sizeof(int));
    int* C = (int*)malloc(size * sizeof(int));
    for (int i = 0; i<size; ++i){A[i] = rand()%100; B[i] = rand()%100;}

    int *dA, *dB, *dC; // these are pointers to int

    // allocate some CUDA memory
    hipMalloc(&dA, size * sizeof(int)); // point to the GPU memory, bitch 
    hipMalloc(&dB, size * sizeof(int)); 
    hipMalloc(&dC, size * sizeof(int)); 

    // Copy input arrays to GPU
    hipMemcpy(dA, A, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size * sizeof(int), hipMemcpyHostToDevice);

    int threads_per_block = 64; //because 64 cores in one SM
    int sm_count = 14;
    int gridDim = min(32*sm_count , size / threads_per_block + 1); // 
    printf("gridDim: %d\n", gridDim);
    int blockDim = threads_per_block;
    // launch kernel
    clock_t start = clock();
    VecAdd<<<gridDim, blockDim>>>(dA, dB, dC);
    clock_t end = clock();
    seeCUDAerror;
    // Copy result back to host
    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);
    print_array(A, B, C);

    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("\nTime taken: %f seconds\n", time_spent);

    return 0;
}

// -------------------------helper functions ----------------------

void print_array(int* A, int* B, int* C){
    printf("A: ");
    for (int i = 0; i<4 ; i++){printf("%d, ", A[i]);};
    printf("\nB: ");
    for (int i = 0; i<4 ; i++){printf("%d, ", B[i]);};
    printf("\nC: ");
    for (int i = 0; i<4 ; i++){printf("%d, ", C[i]);};
}

void seeCUDAerror(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
    };
    hipDeviceSynchronize();   
}