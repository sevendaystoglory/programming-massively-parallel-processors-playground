
#include <hip/hip_runtime.h>
#include <stdio.h>

// we shall understand thread hierarchy
__global__ void VecAdd(int* A, int* B, int * C){
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main(){
    // initialize some variables in CPU memory
    int N = 4; // number of threads
    int A[3] = {1,2,3};
    int B[3] = {10,12,10};
    int C[3];

    int *dA, *dB, *dC; // these are pointers to int

    int size = N*sizeof(int);

    // allocate some CUDA memory
    hipMalloc(&dA, size); // point to the GPU memory, bitch 
    hipMalloc(&dB, size); 
    hipMalloc(&dC, size); 

    // Copy input arrays to GPU
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    // launch kernel
    VecAdd<<<1, N>>>(dA, dB, dC);

    // Copy result back to host
    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);

    for (int i = 0; i<4 ; i++){printf("%d, ", C[i]);};
    return 0;
}